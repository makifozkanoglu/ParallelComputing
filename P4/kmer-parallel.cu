#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/wait.h> 
#include <unistd.h> 
#include <util.h>

__device__ int cuda_strlen(const char* string){
    int length = 0;
    while ( *string && (*string!='\n')){
        length++;
        string++;
    }
    return (length);
}
__device__ int ipow_cuda(int x,int y){

    int result = 1;

    //for(int i = y; i>0; i--) result *= x;
    while (y != 0) {
        result *= x;
        --y;
    }
    return result;
}
__global__ void vector_init(int *x,int len,int element){

    for(int i=0;i<len;i++)
        x[i]=element;

}

/*
__global__ void filter_results(int *r_idxs,
            int *r_line_idx,
            int *r_pat_idx,
            int *r_query_idx,
            int *r_hits,
            int *r_extension_scores, int NUMBER_OF_CUDA_THREADS, int number_of_queries){
    int max,idx;
    for(int ii=0;ii<number_of_queries;ii++){
            max=0,idx=0;
            for(int jj=0;jj<NUMBER_OF_CUDA_THREADS;jj++){
                if (r_extension_scores[jj+ii*NUMBER_OF_CUDA_THREADS]>max){
                    max = r_extension_scores[jj+ii*NUMBER_OF_CUDA_THREADS];
                    idx = jj;
                }
            }
            if (r_hits[idx][ii]>0){
                int query_idx = ii;//r_query_idx[idx][i]
                extension_scores[query_idx]=r_extension_scores[idx][ii];
                idxs[query_idx]=idx;
                line_idxs[query_idx]=r_line_idx[idx][ii];
                pat_idxs[query_idx]=r_pat_idx[idx][ii];
                hits[query_idx]+=r_hits[idx][ii];
            }
        }
}
*/
__device__ char* copy_pattern(char *query_text,int pattern_length){

    char *pattern;
    hipMalloc(&pattern, pattern_length*sizeof(char));
    //for(int i = y; i>0; i--) result *= x;
    for(int i=0;i<pattern_length;i++)
        pattern[i]=query_text[i];
    return pattern;
}
__host__ void kmer_starts(StringList &queries,
                          ResultDict &result,
                          char *reference, int k, 
                          int ref_length,
                          int NUMBER_OF_CUDA_THREADS,int blocksize
                          );
__device__ int get_query_idx(char *query, int k);

__global__ void search(char *c_queries_flattened, int line_queries_used, 
            char *reference, int k, 
            int ref_length, 
            int *r_idxs,
            int *r_line_idx,
            int *r_pat_idx,
            int *r_query_idx,
            int *r_hits,
            int *r_extension_scores, 
            int NUMBER_OF_CUDA_THREADS,
            int number_of_queries,
            int *idxs,
            int *line_idx,
            int *pat_idx,
            int *query_idx,
            int *hits,
            int *extension_scores,
            int *hold_max_idx);
            
__device__  void updateResultDict_cuda(
            int *r_idxs,
            int *r_line_idx,
            int *r_pat_idx,
            int *r_hits,
            int *r_extension_scores,
            int idx, int query_idx, 
            int line_idx,int pat_idx, 
            int extension_score,int tid,int number_of_queries,int *hold);

int main(int argc, char *argv[]){
    clock_t start = clock(), diff;
    //int row_size,col_size,value_size;
    
    if(argc > 4) {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    }

    
    FILE *fp;
    int k;

    //malloc instead of allocating in stack
    char *reference_str = (char*) malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char*) malloc(MAX_READ_LENGTH * sizeof(char));

    /*
    char reference_filename[]= "/home/akif/Parallel/P4/data/ref.txt";
    char read_filename[]= "/home/akif/Parallel/P4/data/reads.txt"; 
    char output_filename[]= "s";k = 3;
    
    */
    int reference_length;
    char *reference_filename, *read_filename, *output_filename;
    reference_filename = argv[1];
    read_filename =argv[2];
    k = atoi(argv[3]);
    output_filename = argv[4];
    


    fp = fopen(reference_filename, "r");
    if (fp == NULL) {
        printf("Could not open file %s!\n",reference_filename);
        return 1;
    }
    
    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }

    substring(reference_str, 0, strlen(reference_str)-1);
    //printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;

    initStringList(&queries, 3);  // initially 3 elements
    
    int success = read_file(read_filename,&queries);
    printf("success%d\n",success);
    //for (int i=0;i<queries.used;i++)//{
    //   printf("Reference str is = %s for %d idx\n", queries.array[i],i);
    //}
    
    reference_length = strlen(reference_str); //Last character is '\n'
    int NUMBER_OF_CUDA_THREADS,blocksize;
    ResultDict result;
    initResultDict(&result, k);
     if(argc == 7) {
        NUMBER_OF_CUDA_THREADS = atoi(argv[5]);
        blocksize=atoi(argv[6]); //* 1024;
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    } else {
        NUMBER_OF_CUDA_THREADS =1024;;
        blocksize=16; //* 1024;
    }
    //int blocksize = 512; // value usually chosen by tuning and hardware constraints
    //int nblocks = NUMBER_OF_CUDA_THREADS / blocksize; // value determine by block size and total work
    //madd<<<nblocks,blocksize>>>mAdd(A,B,C,n);

    printf("Searching\n");
    kmer_starts(queries,result,reference_str,k,reference_length,NUMBER_OF_CUDA_THREADS,blocksize);


    fp = fopen (output_filename, "w+");   
    for (int i=0;i<result.size;i++){
        if(result.hits[i]){
            fprintf(fp, "%s, %d, %d\n",
            result.queries[i],
            result.hits[i],
            result.extension_scores[i]);
        }
    }
    fclose(fp);
    //Free up
    freeResultDict(&result);
    freeStringList(&queries);

    free(reference_str);
    free(read_str);
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("----\nTime taken: %d seconds %d milliseconds\n", msec/1000, msec%1000);

    return 0;

}


__host__ void kmer_starts(StringList &queries,
                          ResultDict &result,
                          char *reference_str, int k, 
                          int reference_length,
                          const int NUMBER_OF_CUDA_THREADS, int blocksize){

    //char **c_line_queries_array; //int c_line_queries_used;
    char *c_reference;
    int size;
    
    size=reference_length * sizeof(char);
    hipMalloc((void**)&c_reference, size);
    hipMemcpy(c_reference, reference_str, size, hipMemcpyHostToDevice);
    


    int number_of_queries = ipow(4,k);
    int *r_idxs,
        *r_line_idx,
        *r_pat_idx,
        *r_query_idx,
        *r_hits,
        *r_extension_scores;
    size=number_of_queries * NUMBER_OF_CUDA_THREADS  * sizeof(int);
    hipMalloc((void**)&r_idxs, size);
    hipMalloc((void**)&r_line_idx, size);
    hipMalloc((void**)&r_pat_idx, size);
    hipMalloc((void**)&r_query_idx, size);
    hipMalloc((void**)&r_hits, size);
    hipMalloc((void**)&r_extension_scores, size);
    int *hold_max_idx;
    hipMalloc((void**)&hold_max_idx, size);

    size=number_of_queries * NUMBER_OF_CUDA_THREADS;
    vector_init<<<1,1>>>(hold_max_idx,size,-1);
    vector_init<<<1,1>>>(r_idxs,size,0);
    vector_init<<<1,1>>>(r_line_idx,size,0);
    vector_init<<<1,1>>>(r_pat_idx,size,0);
    vector_init<<<1,1>>>(r_query_idx,size,0);
    vector_init<<<1,1>>>(r_hits,size,0);
    vector_init<<<1,1>>>(r_extension_scores,size,0);
    
    //char **d_data;
    /*
    hipMalloc(&c_line_queries_array, queries.used*sizeof(char *));
    char **d_temp_data;
    d_temp_data = (char **)malloc(queries.used*sizeof(char *));
    for (int i = 0; i < queries.used; i++){
        hipMalloc(&(d_temp_data[i]), MAX_READ_LENGTH*sizeof(char));
        hipMemcpy(d_temp_data[i], queries.array[i], MAX_READ_LENGTH*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(c_line_queries_array+i, &(d_temp_data[i]), sizeof(char *), hipMemcpyHostToDevice);
    }
    free(d_temp_data);
    
    //////////////////Results//////////////////
    int number_of_queries = ipow(4,k); 
    size = number_of_queries * sizeof(int);
    

    int **r_line_idx;
    //hipMalloc((void**)&r_line_idx, size);
    //hipMemcpy(r_line_idx, result.line_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_line_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_line_idx;
    t_line_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_line_idx[i]), size);
        hipMemcpy(t_line_idx[i], result.line_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_line_idx+i, &(t_line_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
    



    int **r_idxs;
    //hipMalloc((void**)&r_idxs, size);
    //hipMemcpy(r_idxs, result.idxs, size, hipMemcpyHostToDevice);
    hipMalloc(&r_idxs, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    
    int **t_idxs_data;
    t_idxs_data = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_idxs_data[i]), size);
        hipMemcpy(t_idxs_data[i], result.idxs, size, hipMemcpyHostToDevice);
        hipMemcpy(r_idxs+i, &(t_idxs_data[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
     
    
    int **r_pat_idx;
    //hipMalloc((void**)&r_pat_idx, size);
    //hipMemcpy(r_pat_idx, result.pat_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_pat_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));

    int **t_pat_idx;
    t_pat_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_pat_idx[i]), size);
        hipMemcpy(t_pat_idx[i], result.pat_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_pat_idx+i, &(t_pat_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }



    int **r_query_idx;
    //hipMalloc((void**)&r_query_idx, size);
    //hipMemcpy(r_query_idx, result.query_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_query_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_query_idx;
    t_query_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_query_idx[i]), size);
        hipMemcpy(t_query_idx[i], result.query_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_query_idx+i, &(t_query_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }



    int **r_hits;
    //hipMalloc((void**)&r_hits, size);
    //hipMemcpy(r_hits, result.hits, size, hipMemcpyHostToDevice);
    hipMalloc(&r_hits, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_hits;
    t_hits = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_hits[i]), size);
        hipMemcpy(t_hits[i], result.hits, size, hipMemcpyHostToDevice);
        hipMemcpy(r_hits+i, &(t_hits[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
    
    int **r_extension_scores;
    //hipMalloc((void**)&r_extension_scores, size);
    //hipMemcpy(r_extension_scores, result.extension_scores, size, hipMemcpyHostToDevice);
    hipMalloc(&r_extension_scores, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_extension_scores;
    t_extension_scores = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_extension_scores[i]), size);
        hipMemcpy(t_extension_scores[i], result.extension_scores, size, hipMemcpyHostToDevice);
        hipMemcpy(r_extension_scores+i, &(t_extension_scores[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
   
    */
    
    //free(t_extension_scores);
    int nblocks = NUMBER_OF_CUDA_THREADS / blocksize; // value determine by block size and total work
    
    int *idxs;
    
    int *line_idx;
    int *pat_idx;
    int *query_idx;
    int *hits;
    int *extension_scores;
    size = number_of_queries*sizeof(int);
    hipMalloc((void **)&idxs, size);
    hipMalloc((void **)&line_idx, size);
    hipMalloc((void **)&pat_idx, size);
    hipMalloc((void **)&query_idx, size);
    hipMalloc((void **)&hits, size);
    hipMalloc((void **)&extension_scores, size);
    
    char *queries_flattened = (char*) malloc(queries.used * MAX_READ_LENGTH * sizeof(char));
    for (int i=0;i<queries.used;i++){
        //printf("q%d,s:%s\n",i,queries.array[i]);
        for(int j=0;j<MAX_READ_LENGTH;j++)
            queries_flattened[i*MAX_READ_LENGTH+j]=queries.array[i][j];
    }


    char *c_queries_flattened;
    hipMalloc((void **)&c_queries_flattened, queries.used*MAX_READ_LENGTH*sizeof(char));
    hipMemcpy(c_queries_flattened, queries_flattened, queries.used*MAX_READ_LENGTH*sizeof(char), hipMemcpyHostToDevice);

    //////////////////Results//////////////////
    search<<<nblocks, blocksize>>> (c_queries_flattened, 
                                    queries.used, 
                                    c_reference, 
                                    k, 
                                    reference_length, 
                                    r_idxs,
                                    r_line_idx,
                                    r_pat_idx,
                                    r_query_idx,
                                    r_hits,
                                    r_extension_scores,
                                    NUMBER_OF_CUDA_THREADS,
                                    number_of_queries,
                                    idxs,
                                    line_idx,
                                    pat_idx,
                                    query_idx,
                                    hits,
                                    extension_scores,
                                    hold_max_idx
                                    );


    
    
    hipMemcpy(result.idxs, idxs, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.line_idx, line_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.pat_idx, pat_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.query_idx, query_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.hits, hits, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipError_t error = hipMemcpy(result.extension_scores, extension_scores, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
   // SEGFAULT!!!
   if (error != hipSuccess){
        printf("hipMemcpy returned error code %d, line(%d), ErrorString: '%s' \n", error, __LINE__, hipGetErrorString(error));
        //exit(EXIT_FAILURE);
    }

    
    
    //hipFree(c_line_queries_array);
    hipFree(c_reference);
    hipFree(r_idxs);
    hipFree(r_line_idx);
    hipFree(r_pat_idx);
    hipFree(r_query_idx);
    hipFree(r_hits);
    hipFree(r_extension_scores);

    hipFree(idxs);
    hipFree(line_idx);
    hipFree(pat_idx);
    hipFree(query_idx);
    hipFree(hits);
    hipFree(extension_scores);

}



__global__ void search(char *c_queries_flattened, int line_queries_used, 
            char *reference, 
            int k, 
            int ref_length,
            int *r_idxs,
            int *r_line_idx,
            int *r_pat_idx,
            int *r_query_idx,
            int *r_hits,
            int *r_extension_scores,
            int NUMBER_OF_CUDA_THREADS,
            int number_of_queries,
            int *idxs,
            int *line_idxs,
            int *pat_idxs,
            int *query_idxs,
            int *hits,
            int *extension_scores,
            int *hold_max_idx)
{
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    //printf("%d\n",tid);
    int quo=line_queries_used/NUMBER_OF_CUDA_THREADS;
    int remainder=line_queries_used%NUMBER_OF_CUDA_THREADS;
    int batch_size=(remainder<tid)?quo+1:quo;
    int start_idx;
    if(tid<remainder){
        start_idx=tid*(quo+1);
    } else {
        start_idx=tid*quo+remainder;//remainder*(quo+1)+(tid-remainder)*quo;
    }
    

    char pat_text[MAX_READ_LENGTH];
    //hipMalloc(&pat_text, MAX_READ_LENGTH*sizeof(char));
    //for (int line_idx=0; line_idx<line_queries_used; line_idx++){
    for (int l_idx=0; l_idx<batch_size; l_idx++){
        int s_idx=(l_idx+start_idx)*MAX_READ_LENGTH;
        
        //for(int i = y; i>0; i--) result *= x;
        for(int xx=0;xx<MAX_READ_LENGTH;xx++)
            pat_text[xx]=c_queries_flattened[s_idx+xx];

        //char *pat_text = copy_pattern(c_queries_flattened+s_idx, MAX_READ_LENGTH);
        //char *pat_text = c_queries_flattened[l_idx+start_idx];
        //char *pat;
        int pat_len = cuda_strlen(pat_text);

        for(int p_idx=0;p_idx<=pat_len-k;p_idx++){

            int query_idx = get_query_idx(pat_text+p_idx, k);

            for (int idx = 0; idx <= ref_length - k; idx++) {
                int l;/*
        
                /* For current index idx, check for pattern match */
                for (l = 0; l < k; l++)
                    if (reference[idx + l] != pat_text[p_idx+l])
                        break;
        
                if (l == k){ // if pat[0...k-1] = txt[idx, idx+1, ...idx+k-1]
                    //printf("Pattern found at index %d \n", idx);
  
                    int x=idx+k,y=p_idx+k;
                    int extension_score=k;
                    //while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]){
                    while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]&&reference[x]!='\n'&&pat_text[y]!='\n'){
                        //printf("ref%cpattern%c\n",reference[x],pat_text[y]);
                        if(reference[x] == pat_text[y])
                            extension_score++;
                        else break;
                        x++;y++;
                    }
                    x=idx-1,y=p_idx-1;

                    while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]&&reference[x]!='\n'&&pat_text[y]!='\n'){
                    //while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]){
                        if(reference[x] == pat_text[y])
                            extension_score++;
                        else break;
                        x--;y--;
                    }
                  
                    //(r, idx, query_idx, line_idx, pat_idx, extension_score)
                    updateResultDict_cuda(
                                        r_idxs,
                                        r_line_idx,
                                        r_pat_idx,
                                        //r_query_idx,
                                        r_hits,
                                        r_extension_scores,
                                        idx,  
                                        query_idx, 
                                        l_idx, 
                                        p_idx, 
                                        extension_score,
                                        tid,
                                        number_of_queries,
                                        hold_max_idx);

                }
            }
        }

    }
    //hipFree(pat_text);
    __syncthreads();
    //hipDeviceSynchronize();
    
    int idx=0, max=0;
    if(tid==0){
        for(int ii=0;ii<number_of_queries;ii++){
            max=0,idx=0;
            for(int jj=0;jj<NUMBER_OF_CUDA_THREADS;jj++){
                if (r_extension_scores[jj*number_of_queries +ii]>max){
                    max = r_extension_scores[jj*number_of_queries +ii];
                    idx = jj;
                }
            }
            if (r_hits[idx*number_of_queries+ii]>=hits[ii]){
                int query_idx = ii;//r_query_idx[idx][i]
                extension_scores[query_idx]=r_extension_scores[idx*number_of_queries+ii];
                idxs[query_idx]=r_idxs[idx*number_of_queries+ii];//idx;
                line_idxs[query_idx]=r_line_idx[idx*number_of_queries+ii];
                pat_idxs[query_idx]=r_pat_idx[idx*number_of_queries+ii];
                hits[query_idx]=r_hits[idx*number_of_queries+ii];
            }
        }
    }

    __syncthreads();
    
    //insertStringList(result,char *ch='ATC, 4, 4');
}

__device__ int get_query_idx(char *query, int k){
    const char cs[4]={'A','T','G','C'};
    int idx=0;
    int i=0;
    int count=0;
    //printf("wwwwwwwwwww\n");
    for(i=0;i<k;i++){
        int coeff = ipow_cuda(4,k-i-1);
        for(int j=0;j<4;j++){
            //printf("%c",query[i]);
            if (query[i]==cs[j]){
                //printf("yesc:%d   j:%d\n",j,coeff);
                idx+=j*coeff;
                count++;
                break;
            }
        }
    }
    if (count!=k)
        return -1;
    else 
        return idx;
}

__device__  void updateResultDict_cuda(
        int *r_idxs,
        int *r_line_idx,
        int *r_pat_idx,
        int *r_hits,
        int *r_extension_scores,

        int idx, int query_idx, 
        int line_idx,int pat_idx, 
        int extension_score,
        int tid,
        int number_of_queries,int *hold_max_idx)
{   
    int _idx = tid*number_of_queries+query_idx;
    if(idx>hold_max_idx[_idx]&&idx>r_idxs[_idx]){
        r_hits[_idx]++;
        hold_max_idx[_idx] = idx;
        //if(tid==0&&query_idx==63) printf("idx:%d\n");
    }
        
    
    if(r_extension_scores[_idx]<extension_score){
        r_extension_scores[_idx]=extension_score;
        r_idxs[_idx]=idx;
        r_line_idx[_idx]=line_idx;
        r_pat_idx[_idx]=pat_idx;
    }
    
}
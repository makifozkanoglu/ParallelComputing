#include "hip/hip_runtime.h"
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/wait.h> 
#include <unistd.h> 
#include <util.h>
__device__ int cuda_strlen(const char* string){
    int length = 0;
    while (*string++)
        length++;

    return (length);
}
__device__ int ipow_cuda(int x,int y){

    int result = 1;

    //for(int i = y; i>0; i--) result *= x;
    while (y != 0) {
        result *= x;
        --y;
    }
    return result;
}
__host__ void kmer_starts(StringList &queries,
                          ResultDict &result,
                          char *reference, int k, 
                          int ref_length,
                          int NUMBER_OF_CUDA_THREADS
                          );
__device__ int get_query_idx(char *query, int k);

__global__ void search(char **line_queries_array, int line_queries_used, 
            char *reference, int k, 
            int ref_length, int **r_idxs,
            int **r_line_idx,
            int **r_pat_idx,
            int **r_query_idx,
            int **r_hits,
            int **r_extension_scores, int NUMBER_OF_CUDA_THREADS,
            int *idxs,
            int *line_idx,
            int *pat_idx,
            int *query_idx,
            int *hits,
            int *extension_scores);
            
__device__  void updateResultDict_cuda(
            int *r_idxs,
            int *r_line_idx,
            int *r_pat_idx,
            int *r_hits,
            int *r_extension_scores,
            int idx, int query_idx, 
            int line_idx,int pat_idx, 
            int extension_score);

int main(int argc, char *argv[]){
    clock_t start = clock(), diff;
    int row_size,col_size,value_size;
    
        if(argc != 5) {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    }

    
    FILE *fp;
    int k;

    //malloc instead of allocating in stack
    char *reference_str = (char*) malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char*) malloc(MAX_READ_LENGTH * sizeof(char));

    
    char reference_filename[]= "/home/akif/Parallel/P4/data/ref.txt";char read_filename[]= "/home/akif/Parallel/P4/ddata/reads_9216_100bp.txt"; char output_filename[]= "s";k = 3;
    int reference_length;
    /*
    char *reference_filename, *read_filename, *output_filename;
    reference_filename = argv[1];
    read_filename =argv[2];
    k = atoi(argv[3]);
    output_filename = argv[4];
    */



    fp = fopen(reference_filename, "r");
    if (fp == NULL) {
        printf("Could not open file %s!\n",reference_filename);
        return 1;
    }

    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }

    substring(reference_str, 0, strlen(reference_str)-1);
    //printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;

    initStringList(&queries, 3);  // initially 3 elements
      
    int success = read_file(read_filename,&queries);
    //for (int i=0;i<queries.used;i++)//{
    //   printf("Reference str is = %s for %d idx\n", queries.array[i],i);
    //}
      
    reference_length = strlen(reference_str); //Last character is '\n'
    
    ResultDict result;
    initResultDict(&result, k);

    const int NUMBER_OF_CUDA_THREADS = 128; //* 1024;
    //int blocksize = 512; // value usually chosen by tuning and hardware constraints
    //int nblocks = NUMBER_OF_CUDA_THREADS / blocksize; // value determine by block size and total work
    //madd<<<nblocks,blocksize>>>mAdd(A,B,C,n);

    printf("Searching\n");
    kmer_starts(queries,result,reference_str,k,reference_length,NUMBER_OF_CUDA_THREADS);
    
    for (int i=0;i<result.size;i++){
        if(result.hits[i]){
            printf("i:%d,idx:%d %s, hit:%d, extension_score:%d \n",
            i,result.idxs[i], 
            result.queries[i],
            result.hits[i],
            result.extension_scores[i]);
        }
    }
    //Free up
    freeResultDict(&result);
    freeStringList(&queries);

    free(reference_str);
    free(read_str);
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("----\nTime taken: %d seconds %d milliseconds\n", msec/1000, msec%1000);

    return 0;

}

/*
__host__ void kmer_starts(StringList &queries,
                          ResultDict &result,
                          char *reference, int k, 
                          int ref_length,
                          int NUMBER_OF_CUDA_THREADS
                          )
{
    int query_size=ipow(4,k);

    // ... Successfully read from file into "data" ...
    StringList* h_queries = (StringList*)malloc(sizeof(StringList));
    memcpy(h_queries, &queries, 34 * sizeof(StringList);

    for (int i=0; i<numMat; i++){

        hipMalloc(&(h_data[i].elements), rows*cols*sizeof(float));
        hipMemcpy(h_data[i].elements, data[i].elements,  rows*cols*sizeof(float)), hipMemcpyHostToDevice);

     }// matrix data is now on the gpu, now copy the "meta" data to gpu
     Matrix* d_data;
     hipMalloc(&d_data, numMat*sizeof(Matrix)); 
     hipMemcpy(d_data, h_data, numMat*sizeof(Matrix));
}
*/
__host__ void kmer_starts(StringList &queries,
                          ResultDict &result,
                          char *reference_str, int k, 
                          int reference_length,
                          const int NUMBER_OF_CUDA_THREADS){

    char **c_line_queries_array; //int c_line_queries_used;
    char *c_reference;
    int size;

    size=reference_length * sizeof(char);
    hipMalloc((void**)&c_reference, size);
    hipMemcpy(c_reference, reference_str, size, hipMemcpyHostToDevice);
    
    //char **d_data;

    hipMalloc(&c_line_queries_array, queries.used*sizeof(char *));
    char **d_temp_data;
    d_temp_data = (char **)malloc(queries.used*sizeof(char *));
    for (int i = 0; i < queries.used; i++){
        hipMalloc(&(d_temp_data[i]), MAX_READ_LENGTH*sizeof(char));
        hipMemcpy(d_temp_data[i], queries.array[i], MAX_READ_LENGTH*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(c_line_queries_array+i, &(d_temp_data[i]), sizeof(char *), hipMemcpyHostToDevice);
    }
    free(d_temp_data);
    
    //////////////////Results//////////////////
    int number_of_queries = ipow(4,k); 
    size = number_of_queries * sizeof(int);
    

    int **r_line_idx;
    //hipMalloc((void**)&r_line_idx, size);
    //hipMemcpy(r_line_idx, result.line_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_line_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_line_idx;
    t_line_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_line_idx[i]), size);
        hipMemcpy(t_line_idx[i], result.line_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_line_idx+i, &(t_line_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
    



    int **r_idxs;
    //hipMalloc((void**)&r_idxs, size);
    //hipMemcpy(r_idxs, result.idxs, size, hipMemcpyHostToDevice);
    hipMalloc(&r_idxs, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    
    int **t_idxs_data;
    t_idxs_data = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_idxs_data[i]), size);
        hipMemcpy(t_idxs_data[i], result.idxs, size, hipMemcpyHostToDevice);
        hipMemcpy(r_idxs+i, &(t_idxs_data[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
     
    
    int **r_pat_idx;
    //hipMalloc((void**)&r_pat_idx, size);
    //hipMemcpy(r_pat_idx, result.pat_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_pat_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));

    int **t_pat_idx;
    t_pat_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_pat_idx[i]), size);
        hipMemcpy(t_pat_idx[i], result.pat_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_pat_idx+i, &(t_pat_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }



    int **r_query_idx;
    //hipMalloc((void**)&r_query_idx, size);
    //hipMemcpy(r_query_idx, result.query_idx, size, hipMemcpyHostToDevice);
    hipMalloc(&r_query_idx, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_query_idx;
    t_query_idx = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_query_idx[i]), size);
        hipMemcpy(t_query_idx[i], result.query_idx, size, hipMemcpyHostToDevice);
        hipMemcpy(r_query_idx+i, &(t_query_idx[i]), sizeof(int *), hipMemcpyHostToDevice);
    }



    int **r_hits;
    //hipMalloc((void**)&r_hits, size);
    //hipMemcpy(r_hits, result.hits, size, hipMemcpyHostToDevice);
    hipMalloc(&r_hits, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_hits;
    t_hits = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_hits[i]), size);
        hipMemcpy(t_hits[i], result.hits, size, hipMemcpyHostToDevice);
        hipMemcpy(r_hits+i, &(t_hits[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
    
    int **r_extension_scores;
    //hipMalloc((void**)&r_extension_scores, size);
    //hipMemcpy(r_extension_scores, result.extension_scores, size, hipMemcpyHostToDevice);
    hipMalloc(&r_extension_scores, NUMBER_OF_CUDA_THREADS*sizeof(int *));
    int **t_extension_scores;
    t_extension_scores = (int **)malloc(NUMBER_OF_CUDA_THREADS*sizeof(int *));
    for (int i = 0; i < NUMBER_OF_CUDA_THREADS; i++){
        hipMalloc(&(t_extension_scores[i]), size);
        hipMemcpy(t_extension_scores[i], result.extension_scores, size, hipMemcpyHostToDevice);
        hipMemcpy(r_extension_scores+i, &(t_extension_scores[i]), sizeof(int *), hipMemcpyHostToDevice);
    }
   

    
    //free(t_extension_scores);
    int blocksize = NUMBER_OF_CUDA_THREADS; // value usually chosen by tuning and hardware constraints
    int nblocks = NUMBER_OF_CUDA_THREADS / blocksize; // value determine by block size and total work
    int *idxs;
    
    int *line_idx;
    int *pat_idx;
    int *query_idx;
    int *hits;
    int *extension_scores;
    size = number_of_queries*sizeof(int);
    hipMalloc((void **)&idxs, size);
    hipMalloc((void **)&line_idx, size);
    hipMalloc((void **)&pat_idx, size);
    hipMalloc((void **)&query_idx, size);
    hipMalloc((void **)&hits, size);
    hipMalloc((void **)&extension_scores, size);
    //////////////////Results//////////////////
    search<<<nblocks,blocksize>>>(c_line_queries_array, 
                        queries.used, 
                        c_reference, 
                        k, 
                        reference_length, 
                        r_idxs,
                        r_line_idx,
                        r_pat_idx,
                        r_query_idx,
                        r_hits,
                        r_extension_scores,
                        NUMBER_OF_CUDA_THREADS,
                        idxs,
                        line_idx,
                        pat_idx,
                        query_idx,
                        hits,
                        extension_scores
                        );

   /*for (int i=0;i<2;i++){
        if(hits[i]|| 1){
            printf("i:%d,idx:%d %s, hit:%d, extension_score:%d \n",
            i,result.idxs[i], 
            result.queries[i],
            result.hits[i],
        result.extension_scores[i]);
        }
    }*/
    //__syncthreads();
    hipDeviceSynchronize();
    /*
    printf("size1:%lu, size2:%lu",sizeof(result.idxs),sizeof(idxs));
    hipMemcpy(result.idxs, idxs, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.line_idx, line_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.pat_idx, pat_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.query_idx, query_idx, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.hits, hits, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result.extension_scores, extension_scores, number_of_queries*sizeof(int), hipMemcpyDeviceToHost);
    printf("copyyyyy\n");
    for (int i=0;i<21;i++){
        if(result.hits[i]|| 1){
            printf("i:%d,idx:%d %s, hit:%d, extension_score:%d \n",
            i,result.idxs[i], 
            result.queries[i],
            result.hits[i],
        result.extension_scores[i]);
        }
    }
    */
    //syncthreads();
    /*
    int idx=0, max=0;
    for(int i=0;i<5;i++){
        for(int j=0;j<5;j++){
            if (r_extension_scores[j][i]>max){
                max = r_extension_scores[j][i];
                idx = j;
            }
        }
        if (r_hits[idx][i]>0){
            int query_idx = i;//r_query_idx[idx][i]
            result.extension_scores[query_idx]=r_extension_scores[idx][i];
            result.idxs[query_idx]=idx;
            result.line_idx[query_idx]=r_line_idx[idx][i];
            result.pat_idx[query_idx]=r_pat_idx[idx][i];
            result.hits[query_idx]=r_hits[idx][i];
        }
    }
    
    */
    /*
    hipFree(c_line_queries_array);
    hipFree(c_reference);
    hipFree(r_idxs);
    hipFree(r_line_idx);
    hipFree(r_pat_idx);
    hipFree(r_query_idx);
    hipFree(r_hits);
    hipFree(r_extension_scores);
    */

}



__global__ void search(char **line_queries_array, int line_queries_used, 
            char *reference, 
            int k, 
            int ref_length,
            int **r_idxs,
            int **r_line_idx,
            int **r_pat_idx,
            int **r_query_idx,
            int **r_hits,
            int **r_extension_scores,
            int NUMBER_OF_CUDA_THREADS,
            int *idxs,
            int *line_idxs,
            int *pat_idxs,
            int *query_idxs,
            int *hits,
            int *extension_scores)
{
    int tid=blockDim.x*blockIdx.x+threadIdx.x;
    //printf("%d\n",tid);
    int quo=line_queries_used/NUMBER_OF_CUDA_THREADS;
    int remainder=line_queries_used%NUMBER_OF_CUDA_THREADS;
    int batch_size=(remainder<tid)?quo+1:quo;
    int start_idx;
    if(tid<remainder){
        start_idx=tid*(quo+1);
    } else {
        start_idx=tid*quo+remainder;//remainder*(quo+1)+(tid-remainder)*quo;
    }
    //for(int m=0;m<3;m++) printf("%d",m);
    //printf("okk%d %s \n", batch_size,line_queries_array[0]);
    //for (int line_idx=0; line_idx<line_queries_used; line_idx++){
    for (int l_idx=0; l_idx<batch_size; l_idx++){
        char *pat_text = line_queries_array[l_idx+start_idx];
        //char *pat;
        int pat_len = cuda_strlen(pat_text);
        //printf("hbb:%s, %d\n",pat_text, pat_len);
        //printf("line idx=%d, %s \n", line_idx,pat_text);
        for(int p_idx=0;p_idx<=pat_len-k;p_idx++){
            //printf("********************************\n");
            int query_idx = get_query_idx(pat_text+p_idx, k);
            //printf("ok%d %d %s queryidx=%d \n",k, pat_idx,pat_text,query_idx);
            //for (int ts = 0; ts < k; ts++) printf("%c", pat_text[pat_idx+ts]);
            //printf("\n");
            for (int idx = 0; idx <= ref_length - k; idx++) {
                int l;/*
        
                /* For current index idx, check for pattern match */
                for (l = 0; l < k; l++)
                    if (reference[idx + l] != pat_text[p_idx+l])
                        break;
        
                if (l == k){ // if pat[0...k-1] = txt[idx, idx+1, ...idx+k-1]
                    //printf("Pattern found at index %d \n", idx);
  
                    int x=idx+k,y=p_idx+k;
                    int extension_score=k;
                    //while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]){
                    while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]){
                        //printf("ref%cpattern%c\n",reference[x],pat_text[y]);
                        if(reference[x] == pat_text[y])
                            extension_score++;
                        else break;
                        x++;y++;
                    }
                    x=idx-1,y=p_idx-1;

                    while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]){
                    //while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]){
                        if(reference[x] == pat_text[y])
                            extension_score++;
                        else break;
                        x--;y--;
                    }
                    /*
                    if (199>extension_score>150)
                        //printf("extension:%d\n",extension_score);
                    if (extension_score>150){
                        //printf("extension:%d\n refx:%d\n paty=%d\n patter:%s\n",extension_score, x,y,pat_text);
                    }*/
                    //printf("extension:%d\n refx:%d\n paty=%d\n patter:%s\n",extension_score, x,y,pat_text);
                    // calculate query idx and extension score
                    //(r, idx, query_idx, line_idx, pat_idx, extension_score)
                    updateResultDict_cuda(
                                        r_idxs[tid],
                                        r_line_idx[tid],
                                        r_pat_idx[tid],
                                        //r_query_idx,
                                        r_hits[tid],
                                        r_extension_scores[tid],
                                        idx,  
                                        query_idx, 
                                        l_idx, 
                                        p_idx, 
                                        extension_score);

                }
            }
        }

    }
    __syncthreads();
    int idx=0, max=0;
    if(tid==0){
        for(int ii=0;ii<64;ii++){
            max=0,idx=0;
            for(int jj=0;jj<NUMBER_OF_CUDA_THREADS;jj++){
                if (r_extension_scores[jj][ii]>max){
                    max = r_extension_scores[jj][ii];
                    idx = jj;
                }
            }
            if (r_hits[idx][ii]>0){
                int query_idx = ii;//r_query_idx[idx][i]
                extension_scores[query_idx]=r_extension_scores[idx][ii];
                idxs[query_idx]=idx;
                line_idxs[query_idx]=r_line_idx[idx][ii];
                pat_idxs[query_idx]=r_pat_idx[idx][ii];
                hits[query_idx]+=r_hits[idx][ii];
            }
        }
    }
    if(tid==0) //printf("dsaddas");
        for (int i=0;i<64;i++){
            //if(hits[i]|| 1)
                printf("i:%d,idx:%d, hit:%d, extension_score:%d \n",
                i,
                idxs[i], 
                hits[i],
                extension_scores[i]
                );
            
        }
    __syncthreads();
    //insertStringList(result,char *ch='ATC, 4, 4');
}

__device__ int get_query_idx(char *query, int k){
    const char cs[4]={'A','T','G','C'};
    int idx=0;
    int i=0;
    int count=0;
    //printf("wwwwwwwwwww\n");
    for(i=0;i<k;i++){
        int coeff = ipow_cuda(4,k-i-1);
        for(int j=0;j<4;j++){
            //printf("%c",query[i]);
            if (query[i]==cs[j]){
                //printf("yesc:%d   j:%d\n",j,coeff);
                idx+=j*coeff;
                count++;
                break;
            }
        }
        
    }
    //printf("\nwwwwwwwwwww\n");
    if (count!=k)
        return -1;
    else 
        return idx;
}

__device__  void updateResultDict_cuda(
        int *r_idxs,
        int *r_line_idx,
        int *r_pat_idx,
        int *r_hits,
        int *r_extension_scores,
        int idx, int query_idx, 
        int line_idx,int pat_idx, 
        int extension_score)
{   //printf("rexxt%d , %d\n",r_extension_scores[query_idx],extension_score);
    if(r_extension_scores[query_idx]<extension_score){
        r_extension_scores[query_idx]=extension_score;
        r_idxs[query_idx]=idx;
        r_line_idx[query_idx]=line_idx;
        r_pat_idx[query_idx]=pat_idx;
    }
    //printf("rexxt%d , %d\n*********\n",r_extension_scores[query_idx],extension_score);
    r_hits[query_idx]++;
}
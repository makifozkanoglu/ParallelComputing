#include "hip/hip_runtime.h"
#include <util.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/wait.h> 

/*
int check_pattern(const char* pat, const char* txt, int ref_length, int k, int *idx)
{
    //int M = strlen(pat);
    //int N = strlen(txt);
 
    // A loop to slide pat[] one by one 
    for (int i = 0; i <= ref_length - k; i++) {
        int j;
 
        // For current index i, check for pattern match 
        for (j = 0; j < k; j++)
            if (txt[i + j] != pat[j])
                break;
 
        if (j == k){ // if pat[0...k-1] = txt[i, i+1, ...i+k-1]
            printf("Pattern found at index %d \n", i);
            *idx=i;
            return 1;
        }
    }
    return -1;
}
*/
int get_query_idx(char *query, int k){
    const char cs[4]={'A','T','G','C'};
    int idx=0;
    int i=0;
    int count=0;
    //printf("wwwwwwwwwww\n");
    for(i=0;i<k;i++){
        int coeff = ipow(4,k-i-1);
        for(int j=0;j<4;j++){
            //printf("%c",query[i]);
            if (query[i]==cs[j]){
                //printf("yesc:%d   j:%d\n",j,coeff);
                idx+=j*coeff;
                count++;
                break;
            }
        }
        
    }
    //printf("\nwwwwwwwwwww\n");
    if (count!=k)
        return -1;
    else 
        return idx;
}
void search(StringList *line_queries, char *reference, int k, 
            int ref_length, ResultDict *r
            /*, char **idx_ptr, int *idx*/){
    //int idx;
    /*int query_size = ipow(4,k);
    int query_not_exist[query_size];
    for (int i=0;i<query_size;i++)
        query_not_exist[i]=0;*/
    
    int *hold_max_idx=(int *)malloc(r->size*sizeof(int));
    for (int vv=0;vv<r->size;vv++)
        hold_max_idx[0]=-1;

    for (int line_idx=0; line_idx<line_queries->used; line_idx++){
        char *pat_text = line_queries->array[line_idx];
        //char *pat;
        int pat_len = strlen(pat_text);
        //printf("%s, %d\n",pat_text, pat_len);
        //printf("line idx=%d, %s \n", line_idx,pat_text);
        for(int pat_idx=0;pat_idx<pat_len-k;pat_idx++){
            //printf("********************************\n");
            int query_idx = get_query_idx(pat_text+pat_idx, k);


            for (int idx = 0; idx <= ref_length - k; idx++) {
                int l;/*
        
                /* For current index idx, check for pattern match */
                for (l = 0; l < k; l++)
                    if (reference[idx + l] != pat_text[pat_idx+l])
                        break;
        
                if (l == k){ // if pat[0...k-1] = txt[idx, idx+1, ...idx+k-1]
                    int x=idx+k,y=pat_idx+k;
                    int extension_score=k;
                    //while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]){
                    while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]&&reference[x]!='\n'&&pat_text[y]!='\n'){
                        //printf("ref%cpattern%c\n",reference[x],pat_text[y]);
                        if(reference[x] == pat_text[y]&&reference[x]!='\n' )
                            extension_score++;
                        else break;
                        x++;y++;
                    }
                    x=idx-1,y=pat_idx-1;

                    while((0<=x)&&(0<=y)&&reference[x]&&pat_text[y]&&reference[x]!='\n'&&pat_text[y]!='\n'){
                    //while((x<ref_length)&&(y<pat_len)&&reference[x]&&pat_text[y]){
                        if(reference[x] == pat_text[y] )
                            extension_score++;
                        else break;
                        x--;y--;
                    }
                    // calculate query idx and extension score
                    updateResultDict(r, idx, query_idx, line_idx, pat_idx, extension_score, hold_max_idx);

                }
            }
        }

    }

}



int main(int argc, char** argv)
{
    if(argc != 5) {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
    }

    clock_t start = clock(), diff;
    FILE *fp;
    int k;

    //malloc instead of allocating in stack
    char *reference_str = (char*) malloc(MAX_REF_LENGTH * sizeof(char));
    char *read_str = (char*) malloc(MAX_READ_LENGTH * sizeof(char));

    char *reference_filename, *read_filename, *output_filename;
    int reference_length;
    
    reference_filename = argv[1];
    read_filename = argv[2];
    k = atoi(argv[3]);
    output_filename = argv[4];



    fp = fopen(reference_filename, "r");
    if (fp == NULL) {
        printf("Could not open file %s!\n",reference_filename);
        return 1;
    }

    if (fgets(reference_str, MAX_REF_LENGTH, fp) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }

    substring(reference_str, 0, strlen(reference_str)-1);
    //printf("Reference str is = %s\n", reference_str);
    fclose(fp);

    //Read queries
    StringList queries;

    initStringList(&queries, 3);  // initially 3 elements
      
    int success = read_file(read_filename,&queries);
    //for (int i=0;i<queries.used;i++)//{
    //   printf("Reference str is = %s for %d idx\n", queries.array[i],i);
    //}
      
    reference_length = strlen(reference_str); //Last character is '\n'
    
    ResultDict result;
    initResultDict(&result, k);
    
    search(&queries, reference_str, k, reference_length, &result);

    fp = fopen (output_filename, "w+");   
    for (int i=0;i<result.size;i++){
        if(result.hits[i]){
            fprintf(fp, "%s, %d, %d\n",
            result.queries[i],
            result.hits[i],
            result.extension_scores[i]);
        }
    }
    fclose(fp);
    //Free up
    freeResultDict(&result);
    freeStringList(&queries);

    free(reference_str);
    free(read_str);
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("----\nTime taken: %d seconds %d milliseconds\n", msec/1000, msec%1000);

}
